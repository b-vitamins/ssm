#include "hip/hip_runtime.h"
#include "common.h"

#include <ATen/ATen.h>
#include <ATen/OpMathType.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/TypeTraits.h>

namespace ssm {
namespace cuda {

namespace {

template <typename T>
__inline__ __device__ T warp_reduce_sum(T value) {
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    value += __shfl_down_sync(0xffffffff, value, offset);
  }
  return value;
}

template <typename T>
__inline__ __device__ void block_allreduce(
    T thread_sum, T thread_sq_sum, T* shared_sum, T* shared_sq_sum,
    T& total_sum, T& total_sq_sum) {
  const int lane = threadIdx.x % warpSize;
  const int warp = threadIdx.x / warpSize;
  const int num_warps = blockDim.x / warpSize;

  thread_sum = warp_reduce_sum(thread_sum);
  thread_sq_sum = warp_reduce_sum(thread_sq_sum);

  if (lane == 0) {
    shared_sum[warp] = thread_sum;
    shared_sq_sum[warp] = thread_sq_sum;
  }
  __syncthreads();

  total_sum = static_cast<T>(0);
  total_sq_sum = static_cast<T>(0);
  if (warp == 0) {
    total_sum = lane < num_warps ? shared_sum[lane] : static_cast<T>(0);
    total_sq_sum =
        lane < num_warps ? shared_sq_sum[lane] : static_cast<T>(0);

    total_sum = warp_reduce_sum(total_sum);
    total_sq_sum = warp_reduce_sum(total_sq_sum);

    if (lane == 0) {
      shared_sum[0] = total_sum;
      shared_sq_sum[0] = total_sq_sum;
    }
  }
  __syncthreads();

  total_sum = shared_sum[0];
  total_sq_sum = shared_sq_sum[0];
}

template <typename T>
__inline__ __device__ T block_reduce_sum(T value, T* shared) {
  const int lane = threadIdx.x % warpSize;
  const int warp = threadIdx.x / warpSize;
  const int num_warps = blockDim.x / warpSize;

  value = warp_reduce_sum(value);

  if (lane == 0) {
    shared[warp] = value;
  }
  __syncthreads();

  T total = static_cast<T>(0);
  if (warp == 0) {
    total = lane < num_warps ? shared[lane] : static_cast<T>(0);
    total = warp_reduce_sum(total);
    if (lane == 0) {
      shared[0] = total;
    }
  }
  __syncthreads();

  return shared[0];
}

template <typename scalar_t, typename residual_t, typename opmath_t>
__global__ void fused_layer_norm_kernel(
    const scalar_t* __restrict__ x, const residual_t* __restrict__ residual,
    const opmath_t* __restrict__ weight, const opmath_t* __restrict__ bias,
    scalar_t* __restrict__ output, int64_t rows, int64_t hidden, opmath_t eps,
    bool has_residual, bool prenorm, bool is_rms, bool has_bias) {
  const int row = blockIdx.x;
  if (row >= rows) {
    return;
  }

  const int lane_stride = blockDim.x;
  const int64_t offset = static_cast<int64_t>(row) * hidden;
  const scalar_t* __restrict__ x_row = x + offset;
  const residual_t* __restrict__ residual_row =
      has_residual ? residual + offset : nullptr;
  scalar_t* __restrict__ out_row = output + offset;

  opmath_t thread_sum = static_cast<opmath_t>(0);
  opmath_t thread_sq_sum = static_cast<opmath_t>(0);

  for (int64_t col = threadIdx.x; col < hidden; col += lane_stride) {
    opmath_t x_val = static_cast<opmath_t>(x_row[col]);
    opmath_t res_val = has_residual
                           ? static_cast<opmath_t>(residual_row[col])
                           : static_cast<opmath_t>(0);
    opmath_t norm_input = prenorm && has_residual ? x_val + res_val : x_val;
    thread_sum += norm_input;
    thread_sq_sum += norm_input * norm_input;
  }

  extern __shared__ __align__(sizeof(opmath_t)) unsigned char shared_raw[];
  auto* shared = reinterpret_cast<opmath_t*>(shared_raw);
  const int num_warps = blockDim.x / warpSize;
  opmath_t* shared_sum = shared;
  opmath_t* shared_sq_sum = shared + num_warps;

  opmath_t total_sum;
  opmath_t total_sq_sum;
  block_allreduce(thread_sum, thread_sq_sum, shared_sum, shared_sq_sum,
                  total_sum, total_sq_sum);

  opmath_t mean = static_cast<opmath_t>(0);
  opmath_t inv_var = static_cast<opmath_t>(0);
  const opmath_t dim_val = static_cast<opmath_t>(hidden);

  if (is_rms) {
    const opmath_t mean_square = total_sq_sum / dim_val;
    inv_var = static_cast<opmath_t>(1) / sqrt(mean_square + eps);
  } else {
    mean = total_sum / dim_val;
    opmath_t mean_square = total_sq_sum / dim_val;
    opmath_t variance = mean_square - mean * mean;
    variance = variance < static_cast<opmath_t>(0)
                   ? static_cast<opmath_t>(0)
                   : variance;
    inv_var = static_cast<opmath_t>(1) / sqrt(variance + eps);
  }

  for (int64_t col = threadIdx.x; col < hidden; col += lane_stride) {
    opmath_t x_val = static_cast<opmath_t>(x_row[col]);
    opmath_t res_val = has_residual
                           ? static_cast<opmath_t>(residual_row[col])
                           : static_cast<opmath_t>(0);
    opmath_t norm_input = prenorm && has_residual ? x_val + res_val : x_val;
    opmath_t normed = is_rms ? norm_input * inv_var
                             : (norm_input - mean) * inv_var;
    opmath_t scaled = normed * weight[col];
    if (has_bias) {
      scaled += bias[col];
    }
    if (!prenorm && has_residual) {
      scaled += res_val;
    }
    out_row[col] = static_cast<scalar_t>(scaled);
  }
}

inline int64_t next_pow2(int64_t value) {
  int64_t power = 1;
  while (power < value && power < 1024) {
    power <<= 1;
  }
  if (power < 32) {
    power = 32;
  }
  return power;
}

template <typename scalar_t, typename residual_t, typename opmath_t>
void launch_layer_norm_kernel(
    const scalar_t* x_ptr, const residual_t* residual_ptr,
    const opmath_t* weight_ptr, const opmath_t* bias_ptr, scalar_t* out_ptr,
    int64_t rows, int64_t hidden, opmath_t eps, bool has_residual,
    bool prenorm, bool is_rms, bool has_bias, hipStream_t stream) {
  const int threads = static_cast<int>(next_pow2(hidden));
  const int64_t blocks = rows;
  const size_t shared_bytes =
      sizeof(opmath_t) * static_cast<size_t>(threads / warpSize) * 2;
  fused_layer_norm_kernel<scalar_t, residual_t, opmath_t>
      <<<blocks, threads, shared_bytes, stream>>>(
          x_ptr, residual_ptr, weight_ptr, bias_ptr, out_ptr, rows, hidden,
          eps, has_residual, prenorm, is_rms, has_bias);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename scalar_t, typename residual_t, typename opmath_t>
__global__ void fused_layer_norm_backward_kernel(
    const scalar_t* __restrict__ x,
    const residual_t* __restrict__ residual,
    const scalar_t* __restrict__ grad_output,
    const opmath_t* __restrict__ weight,
    opmath_t* __restrict__ grad_weight,
    opmath_t* __restrict__ grad_bias,
    residual_t* __restrict__ grad_residual,
    scalar_t* __restrict__ grad_x, int64_t rows, int64_t hidden, opmath_t eps,
    bool has_residual, bool prenorm, bool is_rms, bool has_bias) {
  const int row = blockIdx.x;
  if (row >= rows) {
    return;
  }

  const int lane_stride = blockDim.x;
  const int64_t offset = static_cast<int64_t>(row) * hidden;
  const scalar_t* __restrict__ x_row = x + offset;
  const residual_t* __restrict__ residual_row =
      has_residual ? residual + offset : nullptr;
  const scalar_t* __restrict__ grad_row = grad_output + offset;
  scalar_t* __restrict__ grad_x_row = grad_x + offset;
  residual_t* __restrict__ grad_res_row =
      has_residual && grad_residual != nullptr ? grad_residual + offset
                                               : nullptr;

  opmath_t thread_sum = static_cast<opmath_t>(0);
  opmath_t thread_sq_sum = static_cast<opmath_t>(0);

  for (int64_t col = threadIdx.x; col < hidden; col += lane_stride) {
    const opmath_t x_val = static_cast<opmath_t>(x_row[col]);
    const opmath_t res_val = has_residual
                                 ? static_cast<opmath_t>(residual_row[col])
                                 : static_cast<opmath_t>(0);
    const opmath_t norm_input =
        (prenorm && has_residual) ? x_val + res_val : x_val;
    thread_sum += norm_input;
    thread_sq_sum += norm_input * norm_input;
  }

  extern __shared__ __align__(sizeof(opmath_t)) unsigned char shared_raw[];
  auto* shared = reinterpret_cast<opmath_t*>(shared_raw);
  const int num_warps = blockDim.x / warpSize;
  opmath_t* shared_sum = shared;
  opmath_t* shared_sq_sum = shared + num_warps;

  const opmath_t total_sum = block_reduce_sum(thread_sum, shared_sum);
  const opmath_t total_sq_sum = block_reduce_sum(thread_sq_sum, shared_sq_sum);

  const opmath_t dim_val = static_cast<opmath_t>(hidden);
  opmath_t mean = static_cast<opmath_t>(0);
  opmath_t inv_var = static_cast<opmath_t>(0);

  if (is_rms) {
    const opmath_t mean_square = total_sq_sum / dim_val;
    inv_var = static_cast<opmath_t>(1) / sqrt(mean_square + eps);
  } else {
    mean = total_sum / dim_val;
    opmath_t mean_square = total_sq_sum / dim_val;
    opmath_t variance = mean_square - mean * mean;
    variance = variance < static_cast<opmath_t>(0) ? static_cast<opmath_t>(0)
                                                   : variance;
    inv_var = static_cast<opmath_t>(1) / sqrt(variance + eps);
  }

  opmath_t* shared_grad_norm_sum = shared + 2 * num_warps;
  opmath_t* shared_grad_norm_dot = shared + 3 * num_warps;
  opmath_t* shared_cross = shared + 4 * num_warps;
  opmath_t* shared_bias = shared + 5 * num_warps;

  opmath_t thread_grad_norm_sum = static_cast<opmath_t>(0);
  opmath_t thread_grad_norm_dot = static_cast<opmath_t>(0);
  opmath_t thread_cross = static_cast<opmath_t>(0);
  opmath_t thread_grad_bias = static_cast<opmath_t>(0);

  for (int64_t col = threadIdx.x; col < hidden; col += lane_stride) {
    const opmath_t x_val = static_cast<opmath_t>(x_row[col]);
    const opmath_t res_val = has_residual
                                 ? static_cast<opmath_t>(residual_row[col])
                                 : static_cast<opmath_t>(0);
    const opmath_t norm_input =
        (prenorm && has_residual) ? x_val + res_val : x_val;
    const opmath_t normed = is_rms ? norm_input * inv_var
                                   : (norm_input - mean) * inv_var;
    const opmath_t grad_pre = static_cast<opmath_t>(grad_row[col]);
    const opmath_t grad_norm = grad_pre * weight[col];

    if (!is_rms) {
      thread_grad_norm_sum += grad_norm;
    }
    thread_grad_norm_dot += grad_norm * normed;
    if (is_rms) {
      thread_cross += grad_norm * norm_input;
    }
    if (has_bias) {
      thread_grad_bias += grad_pre;
    }
  }

  const opmath_t grad_norm_sum = block_reduce_sum(
      is_rms ? static_cast<opmath_t>(0) : thread_grad_norm_sum,
      shared_grad_norm_sum);
  const opmath_t grad_norm_dot =
      block_reduce_sum(thread_grad_norm_dot, shared_grad_norm_dot);
  const opmath_t cross_total = block_reduce_sum(
      is_rms ? thread_cross : static_cast<opmath_t>(0), shared_cross);
  block_reduce_sum(has_bias ? thread_grad_bias : static_cast<opmath_t>(0),
                   shared_bias);

  for (int64_t col = threadIdx.x; col < hidden; col += lane_stride) {
    const opmath_t x_val = static_cast<opmath_t>(x_row[col]);
    const opmath_t res_val = has_residual
                                 ? static_cast<opmath_t>(residual_row[col])
                                 : static_cast<opmath_t>(0);
    const opmath_t norm_input =
        (prenorm && has_residual) ? x_val + res_val : x_val;
    const opmath_t normed = is_rms ? norm_input * inv_var
                                   : (norm_input - mean) * inv_var;
    const opmath_t grad_pre = static_cast<opmath_t>(grad_row[col]);
    const opmath_t grad_norm = grad_pre * weight[col];

    opmath_t grad_norm_input;
    if (is_rms) {
      const opmath_t cross_mean = cross_total / dim_val;
      const opmath_t factor =
          cross_mean * inv_var * inv_var * inv_var;  // inv_std^3
      grad_norm_input = grad_norm * inv_var - norm_input * factor;
    } else {
      const opmath_t numerator =
          grad_norm * dim_val - grad_norm_sum - normed * grad_norm_dot;
      grad_norm_input = numerator * (inv_var / dim_val);
    }

    grad_x_row[col] = static_cast<scalar_t>(grad_norm_input);

    if (has_residual && grad_res_row != nullptr) {
      if (prenorm) {
        grad_res_row[col] = static_cast<residual_t>(grad_norm_input);
      } else {
        grad_res_row[col] = static_cast<residual_t>(grad_pre);
      }
    }

    const opmath_t weight_grad = grad_pre * normed;
    atomicAdd(&grad_weight[col], weight_grad);
    if (has_bias && grad_bias != nullptr) {
      atomicAdd(&grad_bias[col], grad_pre);
    }
  }
}

template <typename scalar_t, typename residual_t, typename opmath_t>
void launch_layer_norm_backward_kernel(
    const scalar_t* x_ptr, const residual_t* residual_ptr,
    const scalar_t* grad_out_ptr, const opmath_t* weight_ptr,
    opmath_t* grad_weight_ptr, opmath_t* grad_bias_ptr,
    residual_t* grad_residual_ptr, scalar_t* grad_x_ptr, int64_t rows,
    int64_t hidden, opmath_t eps, bool has_residual, bool prenorm, bool is_rms,
    bool has_bias, hipStream_t stream) {
  if (rows == 0 || hidden == 0) {
    return;
  }

  const int threads = static_cast<int>(next_pow2(hidden));
  const int64_t blocks = rows;
  const int num_warps = threads / warpSize;
  const size_t shared_bytes =
      sizeof(opmath_t) * static_cast<size_t>(num_warps) * 6;

  fused_layer_norm_backward_kernel<scalar_t, residual_t, opmath_t>
      <<<blocks, threads, shared_bytes, stream>>>(
          x_ptr, residual_ptr, grad_out_ptr, weight_ptr, grad_weight_ptr,
          grad_bias_ptr, grad_residual_ptr, grad_x_ptr, rows, hidden,
          eps, has_residual, prenorm, is_rms, has_bias);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

}  // namespace

at::Tensor fused_layer_norm_cuda(
    const at::Tensor& x, const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias,
    const c10::optional<at::Tensor>& residual, bool is_rms, double eps,
    bool prenorm, bool residual_in_fp32) {
  TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor.");
  TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor.");
  if (bias.has_value()) {
    TORCH_CHECK(bias.value().is_cuda(), "bias must be a CUDA tensor.");
  }
  if (residual.has_value()) {
    TORCH_CHECK(residual.value().is_cuda(), "residual must be a CUDA tensor.");
  }
  c10::cuda::CUDAGuard guard(x.device());

  TORCH_CHECK(x.dim() == 3, "x must have shape (B, L, D).");
  TORCH_CHECK(weight.dim() == 1 && weight.size(0) == x.size(-1),
              "weight must match the last dimension of x.");
  if (bias.has_value()) {
    const auto& bias_tensor = bias.value();
    TORCH_CHECK(bias_tensor.dim() == 1 && bias_tensor.size(0) == x.size(-1),
                "bias must match the last dimension of x.");
  }
  if (residual.has_value()) {
    const auto& residual_tensor = residual.value();
    TORCH_CHECK(residual_tensor.sizes() == x.sizes(),
                "residual must match the shape of x.");
  }

  auto x_contig = x.contiguous();
  auto residual_contig = residual.has_value()
                             ? residual.value().contiguous()
                             : at::Tensor();
  auto weight_contig = weight.contiguous();
  c10::optional<at::Tensor> bias_contig = c10::nullopt;
  if (bias.has_value()) {
    bias_contig = bias.value().contiguous();
  }

  const auto rows = x_contig.size(0) * x_contig.size(1);
  const auto hidden = x_contig.size(2);

  auto output = at::empty_like(x_contig);

  if (rows == 0 || hidden == 0) {
    return output;
  }

  const bool has_residual = residual.has_value();
  const bool has_bias = bias.has_value();

  auto stream = c10::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::kHalf, at::kBFloat16, x_contig.scalar_type(),
      "fused_layer_norm_cuda", [&] {
        using scalar_t_ = scalar_t;
        using opmath_t = at::opmath_type<scalar_t_>;
        constexpr auto opmath_scalar = c10::CppTypeToScalarType<opmath_t>::value;

        at::Tensor weight_compute = weight_contig;
        if (weight_compute.scalar_type() != opmath_scalar) {
          weight_compute = weight_compute.to(
              weight_compute.options().dtype(opmath_scalar));
        }
        if (!weight_compute.is_contiguous()) {
          weight_compute = weight_compute.contiguous();
        }

        at::Tensor bias_compute;
        const opmath_t* bias_ptr = nullptr;
        if (has_bias) {
          bias_compute = bias_contig.value();
          if (bias_compute.scalar_type() != opmath_scalar) {
            bias_compute = bias_compute.to(
                bias_compute.options().dtype(opmath_scalar));
          }
          if (!bias_compute.is_contiguous()) {
            bias_compute = bias_compute.contiguous();
          }
          bias_ptr = bias_compute.data_ptr<opmath_t>();
        }

        const opmath_t* weight_ptr = weight_compute.data_ptr<opmath_t>();

        if (!has_residual) {
          launch_layer_norm_kernel<scalar_t_, scalar_t_, opmath_t>(
              x_contig.data_ptr<scalar_t_>(), nullptr, weight_ptr, bias_ptr,
              output.data_ptr<scalar_t_>(), rows, hidden, static_cast<opmath_t>(eps),
              false, prenorm, is_rms, has_bias, stream);
          return;
        }

        at::Tensor residual_compute = residual_contig;
        if (residual_in_fp32) {
          if (residual_compute.scalar_type() != at::kFloat) {
            residual_compute = residual_compute.to(
                residual_compute.options().dtype(at::kFloat));
          }
          if (!residual_compute.is_contiguous()) {
            residual_compute = residual_compute.contiguous();
          }
          launch_layer_norm_kernel<scalar_t_, float, opmath_t>(
              x_contig.data_ptr<scalar_t_>(),
              residual_compute.data_ptr<float>(), weight_ptr, bias_ptr,
              output.data_ptr<scalar_t_>(), rows, hidden, static_cast<opmath_t>(eps),
              true, prenorm, is_rms, has_bias, stream);
        } else {
          if (residual_compute.scalar_type() != x_contig.scalar_type()) {
            residual_compute =
                residual_compute.to(x_contig.scalar_type());
          }
          if (!residual_compute.is_contiguous()) {
            residual_compute = residual_compute.contiguous();
          }
          launch_layer_norm_kernel<scalar_t_, scalar_t_, opmath_t>(
              x_contig.data_ptr<scalar_t_>(),
              residual_compute.data_ptr<scalar_t_>(), weight_ptr, bias_ptr,
              output.data_ptr<scalar_t_>(), rows, hidden, static_cast<opmath_t>(eps),
              true, prenorm, is_rms, has_bias, stream);
        }
      });

  return output;
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
fused_layer_norm_backward_cuda(
    const at::Tensor& grad_output, const at::Tensor& x,
    const at::Tensor& weight, const c10::optional<at::Tensor>& bias,
    const c10::optional<at::Tensor>& residual, bool is_rms, double eps,
    bool prenorm, bool residual_in_fp32) {
  TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor.");
  TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor.");
  TORCH_CHECK(grad_output.is_cuda(), "grad_output must be a CUDA tensor.");
  if (bias.has_value()) {
    TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor.");
  }
  if (residual.has_value()) {
    TORCH_CHECK(residual->is_cuda(), "residual must be a CUDA tensor.");
  }

  TORCH_CHECK(x.dim() == 3, "x must have shape (B, L, D).");
  TORCH_CHECK(weight.dim() == 1 && weight.size(0) == x.size(-1),
              "weight must match the last dimension of x.");
  TORCH_CHECK(grad_output.sizes() == x.sizes(),
              "grad_output must match the shape of x.");
  if (bias.has_value()) {
    TORCH_CHECK(bias->dim() == 1 && bias->size(0) == x.size(-1),
                "bias must match the last dimension of x.");
  }
  if (residual.has_value()) {
    TORCH_CHECK(residual->sizes() == x.sizes(),
                "residual must match the shape of x.");
  }

  c10::cuda::CUDAGuard guard(x.device());

  auto x_contig = x.contiguous();
  auto grad_out_contig = grad_output.contiguous();
  auto weight_contig = weight.contiguous();
  c10::optional<at::Tensor> bias_contig =
      bias.has_value() ? c10::optional<at::Tensor>(bias->contiguous())
                       : c10::nullopt;
  c10::optional<at::Tensor> residual_contig =
      residual.has_value() ? c10::optional<at::Tensor>(residual->contiguous())
                           : c10::nullopt;

  const auto rows = x_contig.size(0) * x_contig.size(1);
  const auto hidden = x_contig.size(2);

  const bool has_bias = bias.has_value();
  const bool has_residual = residual.has_value();

  if (rows == 0 || hidden == 0) {
    auto grad_x_zero = at::zeros_like(x_contig);
    auto grad_weight_zero = at::zeros_like(weight_contig);
    at::Tensor grad_bias_zero;
    if (has_bias) {
      grad_bias_zero = at::zeros_like(bias_contig.value());
    }
    at::Tensor grad_residual_zero;
    if (has_residual) {
      grad_residual_zero = at::zeros_like(residual_contig.value());
    }
    return std::make_tuple(grad_x_zero.to(x.scalar_type()),
                           grad_weight_zero.to(weight.scalar_type()),
                           has_bias ? grad_bias_zero.to(bias->scalar_type())
                                    : at::Tensor(),
                           has_residual
                               ? grad_residual_zero.to(residual->scalar_type())
                               : at::Tensor());
  }

  auto grad_x = at::empty_like(x_contig);
  at::Tensor grad_weight_out;
  at::Tensor grad_bias_out;
  at::Tensor grad_residual_out;

  auto stream = c10::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::kHalf, at::kBFloat16, x_contig.scalar_type(),
      "fused_layer_norm_backward_cuda", [&] {
        using scalar_t_ = scalar_t;
        using opmath_t = at::opmath_type<scalar_t_>;
        constexpr auto opmath_scalar =
            c10::CppTypeToScalarType<opmath_t>::value;

        at::Tensor weight_compute = weight_contig;
        if (weight_compute.scalar_type() != opmath_scalar) {
          weight_compute = weight_compute.to(
              weight_compute.options().dtype(opmath_scalar));
        }
        if (!weight_compute.is_contiguous()) {
          weight_compute = weight_compute.contiguous();
        }

        at::Tensor grad_weight_local = at::zeros_like(weight_compute);

        opmath_t* grad_bias_ptr = nullptr;
        at::Tensor grad_bias_local;
        if (has_bias) {
          at::Tensor bias_compute = bias_contig.value();
          if (bias_compute.scalar_type() != opmath_scalar) {
            bias_compute = bias_compute.to(
                bias_compute.options().dtype(opmath_scalar));
          }
          if (!bias_compute.is_contiguous()) {
            bias_compute = bias_compute.contiguous();
          }
          grad_bias_local = at::zeros_like(bias_compute);
          grad_bias_ptr = grad_bias_local.data_ptr<opmath_t>();
        }

        scalar_t_* grad_x_ptr = grad_x.data_ptr<scalar_t_>();
        const scalar_t_* x_ptr = x_contig.data_ptr<scalar_t_>();
        const scalar_t_* grad_out_ptr = grad_out_contig.data_ptr<scalar_t_>();
        const opmath_t* weight_ptr = weight_compute.data_ptr<opmath_t>();
        opmath_t* grad_weight_ptr = grad_weight_local.data_ptr<opmath_t>();

        if (!has_residual) {
          launch_layer_norm_backward_kernel<scalar_t_, scalar_t_, opmath_t>(
              x_ptr, nullptr, grad_out_ptr, weight_ptr, grad_weight_ptr,
              grad_bias_ptr, nullptr, grad_x_ptr, rows, hidden,
              static_cast<opmath_t>(eps), false, prenorm, is_rms, has_bias,
              stream);
        } else if (residual_in_fp32) {
          at::Tensor residual_compute = residual_contig.value();
          if (residual_compute.scalar_type() != at::kFloat) {
            residual_compute = residual_compute.to(
                residual_compute.options().dtype(at::kFloat));
          }
          if (!residual_compute.is_contiguous()) {
            residual_compute = residual_compute.contiguous();
          }
          auto grad_res_local =
              at::empty_like(residual_compute, at::MemoryFormat::Contiguous);
          launch_layer_norm_backward_kernel<scalar_t_, float, opmath_t>(
              x_ptr, residual_compute.data_ptr<float>(), grad_out_ptr,
              weight_ptr, grad_weight_ptr, grad_bias_ptr,
              grad_res_local.data_ptr<float>(), grad_x_ptr, rows, hidden,
              static_cast<opmath_t>(eps), true, prenorm, is_rms, has_bias,
              stream);
          grad_residual_out = grad_res_local;
        } else {
          at::Tensor residual_compute = residual_contig.value();
          if (residual_compute.scalar_type() != x_contig.scalar_type()) {
            residual_compute = residual_compute.to(
                residual_compute.options().dtype(x_contig.scalar_type()));
          }
          if (!residual_compute.is_contiguous()) {
            residual_compute = residual_compute.contiguous();
          }
          auto grad_res_local =
              at::empty_like(residual_compute, at::MemoryFormat::Contiguous);
          launch_layer_norm_backward_kernel<scalar_t_, scalar_t_, opmath_t>(
              x_ptr, residual_compute.data_ptr<scalar_t_>(), grad_out_ptr,
              weight_ptr, grad_weight_ptr, grad_bias_ptr,
              grad_res_local.data_ptr<scalar_t_>(), grad_x_ptr, rows, hidden,
              static_cast<opmath_t>(eps), true, prenorm, is_rms, has_bias,
              stream);
          grad_residual_out = grad_res_local;
        }

        grad_weight_out = grad_weight_local;
        if (has_bias) {
          grad_bias_out = grad_bias_local;
        }
      });

  at::Tensor grad_weight_tensor = grad_weight_out.to(weight.scalar_type());
  at::Tensor grad_bias_tensor;
  if (has_bias) {
    grad_bias_tensor = grad_bias_out.to(bias->scalar_type());
  }

  at::Tensor grad_residual_tensor;
  if (has_residual) {
    grad_residual_tensor = grad_residual_out.to(residual->scalar_type());
  }

  return std::make_tuple(grad_x, grad_weight_tensor, grad_bias_tensor,
                         grad_residual_tensor);
}

}  // namespace cuda
}  // namespace ssm

